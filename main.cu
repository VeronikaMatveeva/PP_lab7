#include "hip/hip_runtime.h"
﻿#include ""
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

//лучший результат при соотношении 1 к 5

//#define N 500
//#define div 100

//#define N 1000
//#define div 200

#define N 2000
#define div 400

//#define N 5000
//#define div 1000

//алгоритм перемножения матриц
__global__ void mult_matrix(int* M1, int *M2, int *R, int i) {
	int k = blockIdx.x * (N / div) + threadIdx.x;
	int j = blockIdx.y * (N / div) + threadIdx.y;
	R[k * N + j] += M1[k * N + i] * M2[i * N + j];
}

int main() {

	printf("N = %d, div = %d \n", N, div);

	//заполнение матриц
	int* M1, *M2, *R;
	M1 = new int[N * N];
	M2 = new int[N * N];
	R = new int[N * N];
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			M1[i * N + j] = rand() % 10;
			M2[i * N + j] = rand() % 10;
			R[i * N + j] = 0;
		}
	}

	int* dev_m1, *dev_m2, *dev_r;
	hipError_t cudaStatus;
	hipMalloc((void**)&dev_m1, N * N * sizeof(int));
	hipMalloc((void**)&dev_m2, N * N * sizeof(int));
	hipMalloc((void**)&dev_r, N * N * sizeof(int));

	hipError_t error;

	error = hipMemcpy(dev_m1, M1, N * N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}
	error = hipMemcpy(dev_m2, M2, N * N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}
	error = hipMemcpy(dev_r, R, N * N * sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}

	dim3 grid(div, div);
	dim3 blocks(N / div, N / div);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	//запускаем алгоритм
	for (int i = 0; i < N; ++i) {
		mult_matrix << <grid, blocks >> > (dev_m1, dev_m2, dev_r, i);
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	//записываем время работы
	float time = 0;
	hipEventElapsedTime(&time, start, stop);

	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}

	hipDeviceSynchronize();

	error = hipMemcpy(M1, dev_m1, N * N * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMemcpy(M2, dev_m2, N * N * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}

	error = hipMemcpy(R, dev_r, N * N * sizeof(int), hipMemcpyDeviceToHost);
	if (error != hipSuccess)
	{
		printf("%s\n", hipGetErrorString(error));
	}

	printf("\nTIME: \n");
	printf("%f ms\n", time);

	delete M1;
	delete M2;
	delete R;
	hipFree(dev_m1);
	hipFree(dev_m2);
	hipFree(dev_r);
	return 0;
}